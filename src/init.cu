#include "luaT.h"
#include "THC.h"
#include "THLogAdd.h" /* DEBUG: WTF */

#include <thrust/transform.h>
#include <thrust/reduce.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>

#include "utils.c"
#include "ClassNLLCriterionD.cu"
//#include "cuda.cu"
#include "LSTM12Part2.cu"
#include "events.cu"

LUA_EXTERNC DLL_EXPORT int luaopen_liboxnn(lua_State *L);

int luaopen_liboxnn(lua_State *L)
{

  lua_newtable(L);

  //oxnn_cuda_init(L);
  oxnn_ClassNLLCriterionD_init(L);
  oxnn_LSTM12Part2_init(L);
  oxnn_event_init(L);
  return 1;
}
