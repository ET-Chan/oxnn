#include "utils.h"


/*
	create new event
*/
static int oxnn_event_createEvent(lua_State *L)
{
	hipEvent_t *event = (hipEvent_t *)lua_newuserdata(L, sizeof(hipEvent_t));
	luaL_newmetatable(L, "event.mt");
	lua_setmetatable(L, -2);
	hipError_t  err   = hipEventCreate(event);
	
	if (err!=hipSuccess){
		printf ("error in eventCreate: %s \n", hipGetErrorString(err));
		THError("aborting");	
	}
	return 1;
}


/*
	destroy givent event
*/

static int oxnn_event_destroyEvent(lua_State *L)
{
	hipEvent_t *event = (hipEvent_t *) lua_touserdata(L, 1);
	hipError_t err = hipEventDestroy(*event);
	if (err!=hipSuccess){
		printf ("error in eventDestroy: %s \n", hipGetErrorString(err));
		THError("aborting");	
	}
	return 1;
}

/*
	Provided an event, system will record
	down the current stream onto the stated event.
*/
static int oxnn_event_recordEvent(lua_State *L)
{
	THCState *state = getCutorchState(L);
	hipEvent_t *event = (hipEvent_t *) lua_touserdata(L, 1);
	hipStream_t stream = state->currentStream;
	hipError_t  err = hipEventRecord(*event, stream);
	if (err!=hipSuccess){
		printf ("error in eventRecord: %s \n", hipGetErrorString(err));
		THError("aborting");	
	}
	return 1;
}

/*
	Provided an event idx, system will push
	a synchronization op onto current stream
*/
static int oxnn_event_streamWaitEvent(lua_State *L)
{
	THCState *state = getCutorchState(L);
	hipEvent_t *event = (hipEvent_t *) lua_touserdata(L, 1);
	hipStream_t stream = state->currentStream;
	hipError_t err = hipStreamWaitEvent(stream, *event, 0);
	if (err!=hipSuccess){
		printf ("error in streamWait: %s \n", hipGetErrorString(err));	
	}
	return 1;

}

static const struct luaL_Reg oxnn_event__[] = {
	{"oxnn_event_createEvent", oxnn_event_createEvent},
	{"oxnn_event_destroyEvent",oxnn_event_destroyEvent},
	{"oxnn_event_recordEvent", oxnn_event_recordEvent},
	{"oxnn_event_streamWaitEvent", oxnn_event_streamWaitEvent},
   	{NULL, NULL}

};
/*TODO: implement some clean up functions*/

static void oxnn_event_init(lua_State *L) {

  lua_getglobal(L, "oxnn");
  luaL_register(L, NULL, oxnn_event__);

  lua_pop(L, 1);
}

